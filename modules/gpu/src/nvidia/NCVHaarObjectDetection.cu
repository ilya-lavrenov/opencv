/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

////////////////////////////////////////////////////////////////////////////////
//
// NVIDIA CUDA implementation of Viola-Jones Object Detection Framework
//
// The algorithm and code are explained in the upcoming GPU Computing Gems
// chapter in detail:
//
//   Anton Obukhov, "Haar Classifiers for Object Detection with CUDA"
//   PDF URL placeholder
//   email: aobukhov@nvidia.com, devsupport@nvidia.com
//
// Credits for help with the code to:
// Alexey Mendelenko, Cyril Crassin, and Mikhail Smirnov.
//
////////////////////////////////////////////////////////////////////////////////

#if !defined CUDA_DISABLER

#include <algorithm>
#include <cstdio>

#include "NCV.hpp"
#include "NCVAlg.hpp"
#include "NPP_staging/NPP_staging.hpp"
#include "NCVRuntimeTemplates.hpp"
#include "NCVHaarObjectDetection.hpp"
void applyHaarClassifierAnchorParallelDynTemplate(NcvBool tbInitMaskPositively,
                                                  NcvBool tbCacheTextureIImg,
                                                  NcvBool tbCacheTextureCascade,
                                                  NcvBool tbReadPixelIndexFromVector,
                                                  NcvBool tbDoAtomicCompaction,

                                                  dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                                  Ncv32u *d_IImg, Ncv32u IImgStride,
                                                  Ncv32f *d_weights, Ncv32u weightsStride,
                                                  HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                  Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                  Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                  NcvSize32u anchorsRoi, Ncv32u startStageInc,
                                                  Ncv32u endStageExc, Ncv32f scaleArea)
{
}


void applyHaarClassifierClassifierParallelDynTemplate(NcvBool tbCacheTextureIImg,
                                                      NcvBool tbCacheTextureCascade,
                                                      NcvBool tbDoAtomicCompaction,

                                                      dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                                      Ncv32u *d_IImg, Ncv32u IImgStride,
                                                      Ncv32f *d_weights, Ncv32u weightsStride,
                                                      HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages,
                                                      Ncv32u *d_inMask, Ncv32u *d_outMask,
                                                      Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                                      NcvSize32u anchorsRoi, Ncv32u startStageInc,
                                                      Ncv32u endStageExc, Ncv32f scaleArea)
{
}

void initializeMaskVectorDynTemplate(NcvBool tbMaskByInmask,
                                     NcvBool tbDoAtomicCompaction,

                                     dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                     Ncv32u *d_inMask, Ncv32u *d_outMask,
                                     Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                     NcvSize32u anchorsRoi, Ncv32u step)
{
}


Ncv32u getStageNumWithNotLessThanNclassifiers(Ncv32u N, HaarClassifierCascadeDescriptor &haar,
                                              NCVVector<HaarStage64> &h_HaarStages)
{
    Ncv32u i = 0;
    for (; i<haar.NumStages; i++)
    {
        if (h_HaarStages.ptr()[i].getNumClassifierRootNodes() >= N)
        {
            break;
        }
    }
    return i;
}


NCVStatus ncvApplyHaarClassifierCascade_device(NCVMatrix<Ncv32u> &integral,
                                               NCVMatrix<Ncv32f> &d_weights,
                                               NCVMatrixAlloc<Ncv32u> &d_pixelMask,
                                               Ncv32u &numDetections,
                                               HaarClassifierCascadeDescriptor &haar,
                                               NCVVector<HaarStage64> &h_HaarStages,
                                               NCVVector<HaarStage64> &d_HaarStages,
                                               NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                               NCVVector<HaarFeature64> &d_HaarFeatures,
                                               NcvBool bMaskElements,
                                               NcvSize32u anchorsRoi,
                                               Ncv32u pixelStep,
                                               Ncv32f scaleArea,
                                               INCVMemAllocator &gpuAllocator,
                                               INCVMemAllocator &cpuAllocator,
                                               hipDeviceProp_t &devProp,
                                               hipStream_t cuStream)
{
   
    return NCV_SUCCESS;
}


NCVStatus ncvGrowDetectionsVector_device(NCVVector<Ncv32u> &pixelMask,
                                         Ncv32u numPixelMaskDetections,
                                         NCVVector<NcvRect32u> &hypotheses,
                                         Ncv32u &totalDetections,
                                         Ncv32u totalMaxDetections,
                                         Ncv32u rectWidth,
                                         Ncv32u rectHeight,
                                         Ncv32f curScale,
                                         hipStream_t cuStream)
{
    NCVStatus ncvStat = NCV_SUCCESS;
    return ncvStat;
}


//==============================================================================
//
// Pipeline file
//
//==============================================================================


NCVStatus ncvDetectObjectsMultiScale_device(NCVMatrix<Ncv8u> &d_srcImg,
                                            NcvSize32u srcRoi,
                                            NCVVector<NcvRect32u> &d_dstRects,
                                            Ncv32u &dstNumRects,

                                            HaarClassifierCascadeDescriptor &haar,
                                            NCVVector<HaarStage64> &h_HaarStages,
                                            NCVVector<HaarStage64> &d_HaarStages,
                                            NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                            NCVVector<HaarFeature64> &d_HaarFeatures,

                                            NcvSize32u minObjSize,
                                            Ncv32u minNeighbors,      //default 4
                                            Ncv32f scaleStep,         //default 1.2f
                                            Ncv32u pixelStep,         //default 1
                                            Ncv32u flags,             //default NCVPipeObjDet_Default

                                            INCVMemAllocator &gpuAllocator,
                                            INCVMemAllocator &cpuAllocator,
                                            hipDeviceProp_t &devProp,
                                            hipStream_t cuStream)
{
   
    return NCV_SUCCESS;
}


//==============================================================================
//
// Purely Host code: classifier IO, mock-ups
//
//==============================================================================


NCVStatus ncvApplyHaarClassifierCascade_host(NCVMatrix<Ncv32u> &h_integralImage,
                                             NCVMatrix<Ncv32f> &h_weights,
                                             NCVMatrixAlloc<Ncv32u> &h_pixelMask,
                                             Ncv32u &numDetections,
                                             HaarClassifierCascadeDescriptor &haar,
                                             NCVVector<HaarStage64> &h_HaarStages,
                                             NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                             NCVVector<HaarFeature64> &h_HaarFeatures,
                                             NcvBool bMaskElements,
                                             NcvSize32u anchorsRoi,
                                             Ncv32u pixelStep,
                                             Ncv32f scaleArea)
{
  
    return NCV_SUCCESS;
}


NCVStatus ncvGrowDetectionsVector_host(NCVVector<Ncv32u> &pixelMask,
                                       Ncv32u numPixelMaskDetections,
                                       NCVVector<NcvRect32u> &hypotheses,
                                       Ncv32u &totalDetections,
                                       Ncv32u totalMaxDetections,
                                       Ncv32u rectWidth,
                                       Ncv32u rectHeight,
                                       Ncv32f curScale)
{
      return NCV_SUCCESS;
}


NCVStatus loadFromXML(const std::string &filename,
                      HaarClassifierCascadeDescriptor &haar,
                      std::vector<HaarStage64> &haarStages,
                      std::vector<HaarClassifierNode128> &haarClassifierNodes,
                      std::vector<HaarFeature64> &haarFeatures);


NCVStatus ncvHaarGetClassifierSize(const std::string &filename, Ncv32u &numStages,
                                   Ncv32u &numNodes, Ncv32u &numFeatures)
{
    return NCV_SUCCESS;
}


NCVStatus ncvHaarLoadFromFile_host(const std::string &filename,
                                   HaarClassifierCascadeDescriptor &haar,
                                   NCVVector<HaarStage64> &h_HaarStages,
                                   NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                   NCVVector<HaarFeature64> &h_HaarFeatures)
{
     return NCV_SUCCESS;
}


NCVStatus ncvHaarStoreNVBIN_host(const std::string &filename,
                                 HaarClassifierCascadeDescriptor haar,
                                 NCVVector<HaarStage64> &h_HaarStages,
                                 NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                 NCVVector<HaarFeature64> &h_HaarFeatures)
{
   
    return NCV_SUCCESS;
}

#endif /* CUDA_DISABLER */
